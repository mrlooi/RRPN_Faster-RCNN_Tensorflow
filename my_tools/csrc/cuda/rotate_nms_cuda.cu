#include "hip/hip_runtime.h"

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

//#include <torch/extension.h>

#include <hip/hip_runtime.h>

#include <vector>
#include <iostream>
#include <stdio.h>

#include <cmath>

#define CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    if (error != hipSuccess) { \
      std::cout << hipGetErrorString(error) << std::endl; \
    } \
  } while (0)

#define DIVUP(m,n) ((m) / (n) + ((m) % (n) > 0))
int const threadsPerBlock = sizeof(unsigned long long) * 8;

__device__ inline float trangle_area(float * a, float * b, float * c) {
  return ((a[0] - c[0]) * (b[1] - c[1]) - (a[1] - c[1]) * (b[0] - c[0]))/2.0;
}

__device__ inline float area(float * int_pts, int num_of_inter) {

  float area = 0.0;
  for(int i = 0;i < num_of_inter - 2;i++) {
    area += fabs(trangle_area(int_pts, int_pts + 2 * i + 2, int_pts + 2 * i + 4));
  }
  return area;
}

__device__ inline void reorder_pts(float * int_pts, int num_of_inter)
{

  if(num_of_inter > 0) {

    float center[2];

    center[0] = 0.0;
    center[1] = 0.0;

    for(int i = 0;i < num_of_inter;i++) {
      center[0] += int_pts[2 * i];
      center[1] += int_pts[2 * i + 1];
    }
    center[0] /= num_of_inter;
    center[1] /= num_of_inter;

    float vs[16];
    float v[2];
    float d;
    for(int i = 0;i < num_of_inter;i++) {
      v[0] = int_pts[2 * i]-center[0];
      v[1] = int_pts[2 * i + 1]-center[1];
      d = sqrt(v[0] * v[0] + v[1] * v[1]);
      v[0] = v[0] / d;
      v[1] = v[1] / d;
      if(v[1] < 0) {
        v[0]= - 2 - v[0];
      }
      vs[i] = v[0];
    }

    float temp,tx,ty;
    int j;
    for(int i=1;i<num_of_inter;++i){
      if(vs[i-1]>vs[i]){
        temp = vs[i];
        tx = int_pts[2*i];
        ty = int_pts[2*i+1];
        j=i;
        while(j>0&&vs[j-1]>temp){
          vs[j] = vs[j-1];
          int_pts[j*2] = int_pts[j*2-2];
          int_pts[j*2+1] = int_pts[j*2-1];
          j--;
        }
        vs[j] = temp;
        int_pts[j*2] = tx;
        int_pts[j*2+1] = ty;
      }
    }
  }

}
__device__ inline bool inter2line(float * pts1, float *pts2, int i, int j, float * temp_pts) {

  float a[2];
  float b[2];
  float c[2];
  float d[2];

  float area_abc, area_abd, area_cda, area_cdb;

  a[0] = pts1[2 * i];
  a[1] = pts1[2 * i + 1];

  b[0] = pts1[2 * ((i + 1) % 4)];
  b[1] = pts1[2 * ((i + 1) % 4) + 1];

  c[0] = pts2[2 * j];
  c[1] = pts2[2 * j + 1];

  d[0] = pts2[2 * ((j + 1) % 4)];
  d[1] = pts2[2 * ((j + 1) % 4) + 1];

  area_abc = trangle_area(a, b, c);
  area_abd = trangle_area(a, b, d);

  if(area_abc * area_abd >= 0) {
    return false;
  }

  area_cda = trangle_area(c, d, a);
  area_cdb = area_cda + area_abc - area_abd;

  if (area_cda * area_cdb >= 0) {
    return false;
  }
  float t = area_cda / (area_abd - area_abc);

  float dx = t * (b[0] - a[0]);
  float dy = t * (b[1] - a[1]);
  temp_pts[0] = a[0] + dx;
  temp_pts[1] = a[1] + dy;

  return true;
}

__device__ inline bool in_rect(float pt_x, float pt_y, float * pts) {

  float ab[2];
  float ad[2];
  float ap[2];

  float abab;
  float abap;
  float adad;
  float adap;

  ab[0] = pts[2] - pts[0];
  ab[1] = pts[3] - pts[1];

  ad[0] = pts[6] - pts[0];
  ad[1] = pts[7] - pts[1];

  ap[0] = pt_x - pts[0];
  ap[1] = pt_y - pts[1];

  abab = ab[0] * ab[0] + ab[1] * ab[1];
  abap = ab[0] * ap[0] + ab[1] * ap[1];
  adad = ad[0] * ad[0] + ad[1] * ad[1];
  adap = ad[0] * ap[0] + ad[1] * ap[1];

  return abab >= abap && abap >= 0 && adad >= adap && adap >= 0;
}

__device__ inline int inter_pts(float * pts1, float * pts2, float * int_pts) {

  int num_of_inter = 0;

  for(int i = 0;i < 4;i++) {
    if(in_rect(pts1[2 * i], pts1[2 * i + 1], pts2)) {
      int_pts[num_of_inter * 2] = pts1[2 * i];
      int_pts[num_of_inter * 2 + 1] = pts1[2 * i + 1];
      num_of_inter++;
    }
     if(in_rect(pts2[2 * i], pts2[2 * i + 1], pts1)) {
      int_pts[num_of_inter * 2] = pts2[2 * i];
      int_pts[num_of_inter * 2 + 1] = pts2[2 * i + 1];
      num_of_inter++;
    }
  }

  float temp_pts[2];

  for(int i = 0;i < 4;i++) {
    for(int j = 0;j < 4;j++) {
      bool has_pts = inter2line(pts1, pts2, i, j, temp_pts);
      if(has_pts) {
        int_pts[num_of_inter * 2] = temp_pts[0];
        int_pts[num_of_inter * 2 + 1] = temp_pts[1];
        num_of_inter++;
      }
    }
  }

  return num_of_inter;
}

__device__ inline void convert_region(float * pts , float const * const region) {

  float angle = region[4];
  float a_cos = cos(angle/180.0*3.1415926535);
  float a_sin = sin(angle/180.0*3.1415926535);

  float ctr_x = region[0];
  float ctr_y = region[1];

  float w = region[2];
  float h = region[3];

  float pts_x[4];
  float pts_y[4];

  pts_x[0] = - w / 2;
  pts_x[1] = w / 2;
  pts_x[2] = w / 2;
  pts_x[3] = - w / 2;

  pts_y[0] = - h / 2;
  pts_y[1] = - h / 2;
  pts_y[2] = h / 2;
  pts_y[3] = h / 2;

  for(int i = 0;i < 4;i++) {
    // round to nearest int!!
    pts[7 - 2 * i - 1] = __float2int_rn(a_cos * pts_x[i] - a_sin * pts_y[i] + ctr_x);
    pts[7 - 2 * i] = __float2int_rn(a_sin * pts_x[i] + a_cos * pts_y[i] + ctr_y);
  }

}


__device__ inline float inter(float const * const region1, float const * const region2) {

  float pts1[8];
  float pts2[8];
  float int_pts[16];
  int num_of_inter;

  convert_region(pts1, region1);
  convert_region(pts2, region2);

  num_of_inter = inter_pts(pts1, pts2, int_pts);
  // printf("num_of_inter: %d\n", num_of_inter);

  reorder_pts(int_pts, num_of_inter);

  return area(int_pts, num_of_inter);
}

__device__ inline float devRotateIoU(float const * const region1, float const * const region2) {

  float area1 = region1[2] * region1[3];
  float area2 = region2[2] * region2[3];
  float area_inter = inter(region1, region2);

  float iou = area_inter / (area1 + area2 - area_inter + 1e-8);

  // printf("area1: %.3f, area2: %.3f, area_inter: %.3f, iou: %.3f\n",
  //     area1, area2, area_inter, iou);
  return iou;


}

__global__ void rotate_nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, unsigned long long *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  // cache all column data in this block
  __shared__ float block_boxes[threadsPerBlock * 5];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
  }
  __syncthreads();

  // iterate across each row in this block
  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;  // current row
    const float *cur_box = dev_boxes + cur_box_idx * 5;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;  // if they are the same, skip to next (column)
    }

    // for this row, calculate all ious with each column
    for (i = start; i < col_size; i++) {
      float iou = devRotateIoU(cur_box, block_boxes + i * 5);
      // printf("iou: %.3f\n", iou);
      if (iou > nms_overlap_thresh) {
        t |= 1ULL << i;  // basically storing all overlaps across the columns, hashed into one single ULL index
      }
    }

    const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}


void _set_device(int device_id) {
  int current_device;
  CUDA_CHECK(hipGetDevice(&current_device));
  if (current_device == device_id) {
    return;
  }
  // The call to hipSetDevice must come before any calls to Get, which
  // may perform initialization using the GPU.
  CUDA_CHECK(hipSetDevice(device_id));
}

__global__ void overlaps_kernel(const int N, const int K, const float* dev_boxes,
                           const float * dev_query_boxes, float* dev_overlaps) {

  const int col_start = blockIdx.y;
  const int row_start = blockIdx.x;

  const int row_size =
        min(N - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(K - col_start * threadsPerBlock, threadsPerBlock);


  __shared__ float block_boxes[threadsPerBlock * 5];
  __shared__ float block_query_boxes[threadsPerBlock * 5];
  if (threadIdx.x < col_size) {
    block_query_boxes[threadIdx.x * 5 + 0] =
        dev_query_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
    block_query_boxes[threadIdx.x * 5 + 1] =
        dev_query_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
    block_query_boxes[threadIdx.x * 5 + 2] =
        dev_query_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
    block_query_boxes[threadIdx.x * 5 + 3] =
        dev_query_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
    block_query_boxes[threadIdx.x * 5 + 4] =
        dev_query_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
  }

  if (threadIdx.x < row_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        dev_boxes[(threadsPerBlock * row_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        dev_boxes[(threadsPerBlock * row_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        dev_boxes[(threadsPerBlock * row_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        dev_boxes[(threadsPerBlock * row_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        dev_boxes[(threadsPerBlock * row_start + threadIdx.x) * 5 + 4];
  }

  __syncthreads();

  if (threadIdx.x < row_size) {

    for(int i = 0;i < col_size; i++) {
      int offset = row_start*threadsPerBlock * K + col_start*threadsPerBlock + threadIdx.x*K+ i ;
      dev_overlaps[offset] = devRotateIoU(block_boxes + threadIdx.x * 5, block_query_boxes + i * 5);
    }
  }
}


void _overlaps_launcher(float* overlaps, const float* boxes, const float* query_boxes, int n, int k, hipStream_t stream)
{

//  _set_device(device_id);

  float* overlaps_dev = NULL;
  float* boxes_dev = NULL;
  float* query_boxes_dev = NULL;

  CUDA_CHECK(hipMalloc(&boxes_dev,
                        n * 5 * sizeof(float)));

  CUDA_CHECK(hipMemcpy(boxes_dev,
                        boxes,
                        n * 5 * sizeof(float),
                        hipMemcpyHostToDevice));

  CUDA_CHECK(hipMalloc(&query_boxes_dev,
                        k * 5 * sizeof(float)));

  CUDA_CHECK(hipMemcpy(query_boxes_dev,
                        query_boxes,
                        k * 5 * sizeof(float),
                        hipMemcpyHostToDevice));

  CUDA_CHECK(hipMalloc(&overlaps_dev,
                        n * k * sizeof(float)));

  dim3 blocks(DIVUP(n, threadsPerBlock),
              DIVUP(k, threadsPerBlock));

  dim3 threads(threadsPerBlock);

  overlaps_kernel<<<blocks, threads, 0, stream>>>(n, k,
                                    boxes_dev,
                                    query_boxes_dev,
                                    overlaps_dev);

  CUDA_CHECK(hipMemcpy(overlaps,
                        overlaps_dev,
                        n * k * sizeof(float),
                        hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(overlaps_dev));
  CUDA_CHECK(hipFree(boxes_dev));
  CUDA_CHECK(hipFree(query_boxes_dev));
}

void _rotate_nms_launcher(long* keep_out, int* num_out, const float* boxes_host, int boxes_num,
          int boxes_dim, float nms_overlap_thresh, hipStream_t stream)
{
  /**
  Inputs:
  boxes_host: N,5  (xc,yc,w,h,angle)  ASSUMES already sorted
  boxes_num: N
  boxes_dim: 5
  nms_overlap_thresh: 0-1 e.g. 0.7

  Outputs:
  keep_out: N  (i.e. stores indices of valid boxes_host)
  num_out: total count of valid indices

  */
//  _set_device(device_id);

  float* boxes_dev = NULL;
  unsigned long long* mask_dev = NULL;

  const int col_blocks = DIVUP(boxes_num, threadsPerBlock);

  CUDA_CHECK(hipMalloc(&boxes_dev,
                        boxes_num * boxes_dim * sizeof(float)));
  CUDA_CHECK(hipMemcpy(boxes_dev,
                        boxes_host,
                        boxes_num * boxes_dim * sizeof(float),
                        hipMemcpyHostToDevice));


  // Get the IoUs between each element in the array (N**2 operation)
  // then store all the overlap results in the N*col_blocks array (mask_dev).
  // col_blocks represents the total number of column blocks (blockDim.x) made for the kernel computation
  // Each column block will store a hash of the iou overlaps between each column and row in the block. The hash is a ULL of bit overlaps between one row and all columns in the block
  // then copy the results to host code
  // Each result row is a col_block array, which contains all the iou overlap bool (as a hash) per column block.
  // Loop through the col_block array to aggregate all iou overlap results for that row
  CUDA_CHECK(hipMalloc(&mask_dev,
                        boxes_num * col_blocks * sizeof(unsigned long long)));

  dim3 blocks(col_blocks, col_blocks);
  dim3 threads(threadsPerBlock);

  rotate_nms_kernel<<<blocks, threads, 0, stream>>>(boxes_num,
                                  nms_overlap_thresh,
                                  boxes_dev,
                                  mask_dev);
  hipDeviceSynchronize();

  std::vector<unsigned long long> mask_host(boxes_num * col_blocks);
  CUDA_CHECK(hipMemcpy(&mask_host[0],
                        mask_dev,
                        sizeof(unsigned long long) * boxes_num * col_blocks,
                        hipMemcpyDeviceToHost));

  std::vector<unsigned long long> remv(col_blocks);
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

  int num_to_keep = 0;
  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / threadsPerBlock;  // get column block
    int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {  // if not zero i.e. no overlap
      keep_out[num_to_keep++] = long(i);
      unsigned long long *p = &mask_host[0] + i * col_blocks;

      // Loop through the col_block array to aggregate all iou overlap results for that box
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }
  *num_out = num_to_keep;

  CUDA_CHECK(hipFree(boxes_dev));
  CUDA_CHECK(hipFree(mask_dev));
}


at::Tensor rotate_nms_cuda(
    const at::Tensor& r_boxes, const float nms_threshold, const int max_output
)
{

  int boxes_num = r_boxes.size(0);
  int channels = r_boxes.size(1);

  at::Tensor keep = at::zeros({boxes_num}, r_boxes.options().dtype(at::kLong).device(at::kCPU));

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int num_to_keep = 0;
  _rotate_nms_launcher(keep.data<long>(), &num_to_keep, r_boxes.contiguous().data<float>(), boxes_num,
          channels, nms_threshold, stream);
//  AveragedistanceBackwardLaucher(
//    grad.contiguous().data<float>(), bottom_diff.contiguous().data<float>(),
//    batch_size, channels, output.data<float>(), stream
//  );
  THCudaCheck(hipGetLastError());

  // TODO improve this part
//  printf("GPU: num_to_keep: %d\n", num_to_keep);
  return keep.narrow(/*dim=*/0, /*start=*/0, /*length=*/num_to_keep).to(
      r_boxes.device(), keep.scalar_type());

}
