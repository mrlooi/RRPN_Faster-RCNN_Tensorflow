#include "hip/hip_runtime.h"

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

//#include <torch/extension.h>

#include <hip/hip_runtime.h>

#include <vector>
#include <iostream>
#include <stdio.h>

#include <cmath>

#include "rotate_rect_ops.h"

#define CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    if (error != hipSuccess) { \
      std::cout << hipGetErrorString(error) << std::endl; \
    } \
  } while (0)

#define DIVUP(m,n) ((m) / (n) + ((m) % (n) > 0))
int const threadsPerBlock = 512;


template <typename T>
__device__ inline float devRotateIoU(T const * const region1, T const * const region2) {

  return computeRectIoU(region1, region2);
}

__global__ void rotate_nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, unsigned long long *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  // cache all column data in this block
  __shared__ float block_boxes[threadsPerBlock * 5];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
  }
  __syncthreads();

  // iterate across each row in this block
  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;  // current row
    const float *cur_box = dev_boxes + cur_box_idx * 5;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;  // if they are the same, skip to next (column)
    }

    // for this row, calculate all ious with each column
    for (i = start; i < col_size; i++) {
      float iou = devRotateIoU(cur_box, block_boxes + i * 5);
      // printf("iou: %.3f\n", iou);
      if (iou > nms_overlap_thresh) {
        t |= 1ULL << i;  // basically storing all overlaps across the columns, hashed into one single ULL index
      }
    }

    const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

__global__ void overlaps_kernel(const int N, const int K, const float* dev_boxes,
                           const float * dev_query_boxes, float* dev_overlaps) {

  const int col_start = blockIdx.y;
  const int row_start = blockIdx.x;

  const int row_size =
        min(N - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(K - col_start * threadsPerBlock, threadsPerBlock);


  __shared__ float block_boxes[threadsPerBlock * 5];
  __shared__ float block_query_boxes[threadsPerBlock * 5];
  if (threadIdx.x < col_size) {
    block_query_boxes[threadIdx.x * 5 + 0] =
        dev_query_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
    block_query_boxes[threadIdx.x * 5 + 1] =
        dev_query_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
    block_query_boxes[threadIdx.x * 5 + 2] =
        dev_query_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
    block_query_boxes[threadIdx.x * 5 + 3] =
        dev_query_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
    block_query_boxes[threadIdx.x * 5 + 4] =
        dev_query_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
  }

  if (threadIdx.x < row_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        dev_boxes[(threadsPerBlock * row_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        dev_boxes[(threadsPerBlock * row_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        dev_boxes[(threadsPerBlock * row_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        dev_boxes[(threadsPerBlock * row_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        dev_boxes[(threadsPerBlock * row_start + threadIdx.x) * 5 + 4];
  }

  __syncthreads();

  if (threadIdx.x < row_size) {

    for(int i = 0;i < col_size; i++) {
      int offset = row_start*threadsPerBlock * K + col_start*threadsPerBlock + threadIdx.x*K+ i ;
      dev_overlaps[offset] = devRotateIoU(block_boxes + threadIdx.x * 5, block_query_boxes + i * 5);
    }
  }
}


void _iou_matrix_launcher(float* overlaps, const float* boxes, const float* query_boxes,
        int n, int k, hipStream_t stream)
{

  dim3 blocks(DIVUP(n, threadsPerBlock),
              DIVUP(k, threadsPerBlock));

  dim3 threads(threadsPerBlock);

  overlaps_kernel<<<blocks, threads, 0, stream>>>(n, k,
                                    boxes,
                                    query_boxes,
                                    overlaps);
  hipDeviceSynchronize();

}

void _rotate_nms_launcher(long* keep_out, int* num_out, const float* boxes, int boxes_num,
          int boxes_dim, float nms_overlap_thresh, hipStream_t stream)
{
  /**
  Inputs:
  boxes: N,5  (xc,yc,w,h,angle)  ASSUMES already sorted
  boxes_num: N
  boxes_dim: 5
  nms_overlap_thresh: 0-1 e.g. 0.7

  Outputs:
  keep_out: N  (i.e. stores indices of valid boxes)
  num_out: total count of valid indices

  */

  const int col_blocks = DIVUP(boxes_num, threadsPerBlock);

  unsigned long long* mask_dev = NULL;
  // Get the IoUs between each element in the array (N**2 operation)
  // then store all the overlap results in the N*col_blocks array (mask_dev).
  // col_blocks represents the total number of column blocks (blockDim.x) made for the kernel computation
  // Each column block will store a hash of the iou overlaps between each column and row in the block. The hash is a ULL of bit overlaps between one row and all columns in the block
  // then copy the results to host code
  // Each result row is a col_block array, which contains all the iou overlap bool (as a hash) per column block.
  // Loop through the col_block array to aggregate all iou overlap results for that row
  CUDA_CHECK(hipMalloc(&mask_dev,
                        boxes_num * col_blocks * sizeof(unsigned long long)));

  dim3 blocks(col_blocks, col_blocks);
  dim3 threads(threadsPerBlock);

  rotate_nms_kernel<<<blocks, threads, 0, stream>>>(boxes_num,
                                  nms_overlap_thresh,
                                  boxes,
                                  mask_dev);
  hipDeviceSynchronize();

  std::vector<unsigned long long> mask_host(boxes_num * col_blocks);
  CUDA_CHECK(hipMemcpy(&mask_host[0],
                        mask_dev,
                        sizeof(unsigned long long) * boxes_num * col_blocks,
                        hipMemcpyDeviceToHost));

  std::vector<unsigned long long> remv(col_blocks);
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

  int num_to_keep = 0;
  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / threadsPerBlock;  // get column block
    int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {  // if not zero i.e. no overlap
      keep_out[num_to_keep++] = long(i);
      unsigned long long *p = &mask_host[0] + i * col_blocks;

      // Loop through the col_block array to aggregate all iou overlap results for that box
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }
  *num_out = num_to_keep;

  CUDA_CHECK(hipFree(mask_dev));
}


at::Tensor rotate_nms_cuda(
    const at::Tensor& r_boxes, const float nms_threshold
)
{
  int boxes_num = r_boxes.size(0);
  int channels = r_boxes.size(1);

  at::Tensor keep = at::zeros({boxes_num}, r_boxes.options().dtype(at::kLong).device(at::kCPU));

  if (boxes_num == 0)
    return keep;

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int num_to_keep = 0;
  _rotate_nms_launcher(keep.data<long>(), &num_to_keep, r_boxes.contiguous().data<float>(), boxes_num,
          channels, nms_threshold, stream);
//  AveragedistanceBackwardLaucher(
//    grad.contiguous().data<float>(), bottom_diff.contiguous().data<float>(),
//    batch_size, channels, output.data<float>(), stream
//  );
  THCudaCheck(hipGetLastError());

  // TODO improve this part
//  printf("GPU: num_to_keep: %d\n", num_to_keep);
  return keep.narrow(/*dim=*/0, /*start=*/0, /*length=*/num_to_keep).to(
      r_boxes.device(), keep.scalar_type());

}

at::Tensor rotate_iou_matrix_cuda(
    const at::Tensor& r_boxes1, const at::Tensor& r_boxes2
)
{
  int N = r_boxes1.size(0);
  int M = r_boxes2.size(0);

  at::Tensor iou_matrix = at::zeros({N, M}, r_boxes1.options());

  if (N == 0 || M == 0)
    return iou_matrix;

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  _iou_matrix_launcher(iou_matrix.data<float>(), r_boxes1.contiguous().data<float>(),
        r_boxes2.contiguous().data<float>(), N, M, stream);

  THCudaCheck(hipGetLastError());

  return iou_matrix;
}